#include "hip/hip_runtime.h"
#include "./fwd.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

MyErrCode checkDevice(int argc, char** argv)
{
    printf("%s Starting...\n", argv[0]);
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n", static_cast<int>(error_id),
               hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    if (device_count == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("Detected %d CUDA Capable device(s)\n", device_count);
    }
    int dev, driver_version = 0, runtime_version = 0;
    dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, dev);
    printf("Device %d: \"%s\"\n", dev, device_prop.name);
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);
    printf(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driver_version / 1000,
           (driver_version % 100) / 10, runtime_version / 1000, (runtime_version % 100) / 10);
    printf(" CUDA Capability Major/Minor version number: %d.%d\n", device_prop.major,
           device_prop.minor);
    printf(" Total amount of global memory: %.2f GBytes (%llu bytes)\n",
           static_cast<float>(device_prop.totalGlobalMem) / (pow(1024.0, 3)),
           static_cast<unsigned long long>(device_prop.totalGlobalMem));
    printf(" GPU Clock rate: %.0f MHz (%0.2f GHz)\n", device_prop.clockRate * 1e-3f,
           device_prop.clockRate * 1e-6f);
    printf(" Memory Clock rate: %.0f Mhz\n", device_prop.memoryClockRate * 1e-3f);
    printf(" Memory Bus Width: %d-bit\n", device_prop.memoryBusWidth);
    if (device_prop.l2CacheSize) {
        printf(" L2 Cache Size: %d bytes\n", device_prop.l2CacheSize);
    }
    printf(
        " Max Texture Dimension Size (x,y,z) "
        " 1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
        device_prop.maxTexture1D, device_prop.maxTexture2D[0], device_prop.maxTexture2D[1],
        device_prop.maxTexture3D[0], device_prop.maxTexture3D[1], device_prop.maxTexture3D[2]);
    printf(" Max Layered Texture Size (dim) x layers\n  1D = (%d) x % d\n  2D = (%d, % d) x % d\n ",
           device_prop.maxTexture1DLayered[0], device_prop.maxTexture1DLayered[1],
           device_prop.maxTexture2DLayered[0], device_prop.maxTexture2DLayered[1],
           device_prop.maxTexture2DLayered[2]);
    printf(" Total amount of constant memory: %zu bytes\n", device_prop.totalConstMem);
    printf(" Total amount of shared memory per block: %zu bytes\n", device_prop.sharedMemPerBlock);
    printf(" Total number of registers available per block: %d\n", device_prop.regsPerBlock);
    printf(" Warp size: %d\n", device_prop.warpSize);
    printf(" Multi Processor Count: %d\n", device_prop.multiProcessorCount);
    printf(" Maximum number of threads per multiprocessor: %d\n",
           device_prop.maxThreadsPerMultiProcessor);
    printf(" Maximum number of threads per block: %d\n", device_prop.maxThreadsPerBlock);
    printf(" Maximum sizes of each dimension of a block: %d x %d x %d\n",
           device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1],
           device_prop.maxThreadsDim[2]);
    printf(" Maximum sizes of each dimension of a grid: %d x %d x %d\n", device_prop.maxGridSize[0],
           device_prop.maxGridSize[1], device_prop.maxGridSize[2]);
    printf(" Maximum memory pitch: %zu bytes\n", device_prop.memPitch);

    return MyErrCode::kOk;
}