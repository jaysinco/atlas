#include "hip/hip_runtime.h"
#include "./common.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float dev_data[1];

__global__ void checkGlobalVariable()
{
    // display the original value
    printf("Device: the value of the global variable is %f\n", dev_data[0]);

    // alter the value
    dev_data[0] += 2.0f;
}

int globalVariable(int argc, char** argv)
{
    // initialize the global variable
    float value = 3.14f;
    float* d_ptr;
    CHECK(hipGetSymbolAddress((void**)&d_ptr, dev_data));
    CHECK(hipMemcpy(d_ptr, &value, sizeof(float), hipMemcpyHostToDevice));
    printf("Host:   copied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1, 1>>>();

    // copy the global variable back to the host
    CHECK(hipMemcpy(&value, d_ptr, sizeof(float), hipMemcpyDeviceToHost));
    printf("Host:   the value changed by the kernel to %f\n", value);

    CHECK(hipDeviceSynchronize());
    return EXIT_SUCCESS;
}