#include "hip/hip_runtime.h"
#include "./common.cuh"
#include <hip/hip_runtime.h>

__global__ void testPrint()
{
    printf("Hello World from GPU1!\n");
    printf("Hello World from GPU2!\n");
    printf("Hello World from GPU3!\n");
}

int helloWorld(int argc, char** argv)
{
    testPrint<<<1, 1>>>();
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    return 0;
}
