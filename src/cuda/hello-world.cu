#include "hip/hip_runtime.h"
#include "./common.cuh"
#include <hip/hip_runtime.h>

__global__ void testPrint()
{
    printf("Hello World from GPU1!\n");
    printf("Hello World from GPU2!\n");
    printf("Hello World from GPU3!\n");
}

MyErrCode helloWorld(int argc, char** argv)
{
    testPrint<<<1, 1>>>();
    CHECK_CUDA(hipPeekAtLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    return MyErrCode::kOk;
}
