#include "./common.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define MY_PI 3.14159265358979323846

void print_2d(char const* header, hipfftComplex* cs, int nx, int ny)
{
    std::cout << header << std::endl;
    for (int x = 0; x < nx; ++x) {
        for (int y = 0; y < ny; ++y) {
            hipfftComplex& it = cs[y + x * ny];
            std::cout << it.x << "+" << it.y << "i"
                      << " ";
        }
        std::cout << "\n";
    }
    std::cout << std::endl;
}

int cufft_test(int argc, char** argv)
{
    int nx = 1920;
    int ny = 1080;
    int ns = nx * ny;
    hipfftComplex *dComplex, *complex;

    // Allocate device memory
    complex = static_cast<hipfftComplex*>(malloc(sizeof(hipfftComplex) * ns));
    CHECK(hipMalloc(&dComplex, sizeof(hipfftComplex) * ns));

    // Input Generation
    for (int x = 0; x < nx; ++x) {
        for (int y = 0; y < ny; ++y) {
            hipfftComplex c;
            c.x = x + y;
            c.y = 0;
            complex[y + x * ny] = c;
        }
    }
    // print_2d("=== INPUT ===", complex, nx, ny);

    // Setup the cuFFT plan
    hipfftHandle plan = 0;
    CHECK_CUFFT(hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C));

    // Transfer inputs into device memory
    CHECK(hipMemcpy(dComplex, complex, sizeof(hipfftComplex) * ns, hipMemcpyHostToDevice));

    // warm up
    warmUpGpu();

    // Execute a complex-to-complex 1D FFT
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start, 0));

    CHECK_CUFFT(hipfftExecC2C(plan, dComplex, dComplex, HIPFFT_FORWARD));

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop););

    float time_ms;
    CHECK(hipEventElapsedTime(&time_ms, start, stop));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    ILOG("fft2 {}x{}: {:.3f}ms", nx, ny, time_ms);

    // Retrieve the results into host memory
    CHECK(hipMemcpy(complex, dComplex, sizeof(hipfftComplex) * ns, hipMemcpyDeviceToHost));

    // print_2d("=== OUTPUT ===", complex, nx, ny);

    free(complex);
    CHECK(hipFree(dComplex));
    CHECK_CUFFT(hipfftDestroy(plan));

    return 0;
}