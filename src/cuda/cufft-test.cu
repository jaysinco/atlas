#include "./common.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define MY_PI 3.14159265358979323846

static void print2d(char const* header, hipfftComplex* cs, int nx, int ny)
{
    std::cout << header << std::endl;
    for (int x = 0; x < nx; ++x) {
        for (int y = 0; y < ny; ++y) {
            hipfftComplex& it = cs[y + x * ny];
            std::cout << it.x << "+" << it.y << "i"
                      << " ";
        }
        std::cout << "\n";
    }
    std::cout << std::endl;
}

int cufftTest(int argc, char** argv)
{
    int nx = 1920;
    int ny = 1080;
    int ns = nx * ny;
    hipfftComplex *d_complex, *complex;

    // Allocate device memory
    complex = static_cast<hipfftComplex*>(malloc(sizeof(hipfftComplex) * ns));
    CHECK(hipMalloc(&d_complex, sizeof(hipfftComplex) * ns));

    // Input Generation
    for (int x = 0; x < nx; ++x) {
        for (int y = 0; y < ny; ++y) {
            hipfftComplex c;
            c.x = x + y;
            c.y = 0;
            complex[y + x * ny] = c;
        }
    }
    print2d("=== INPUT ===", complex, nx, ny);

    // Setup the cuFFT plan
    hipfftHandle plan = 0;
    CHECK_CUFFT(hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C));

    // Transfer inputs into device memory
    CHECK(hipMemcpy(d_complex, complex, sizeof(hipfftComplex) * ns, hipMemcpyHostToDevice));

    // warm up
    warmUpGpu();

    // Execute a complex-to-complex 1D FFT
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start, nullptr));

    CHECK_CUFFT(hipfftExecC2C(plan, d_complex, d_complex, HIPFFT_FORWARD));

    CHECK(hipEventRecord(stop, nullptr));
    CHECK(hipEventSynchronize(stop););

    float time_ms;
    CHECK(hipEventElapsedTime(&time_ms, start, stop));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    ILOG("fft2 {}x{}: {:.3f}ms", nx, ny, time_ms);

    // Retrieve the results into host memory
    CHECK(hipMemcpy(complex, d_complex, sizeof(hipfftComplex) * ns, hipMemcpyDeviceToHost));

    // print_2d("=== OUTPUT ===", complex, nx, ny);

    free(complex);
    CHECK(hipFree(d_complex));
    CHECK_CUFFT(hipfftDestroy(plan));

    return 0;
}