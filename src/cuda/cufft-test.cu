#include "./common.cuh"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define MY_PI 3.14159265358979323846

MyErrCode cufftTest(int argc, char** argv)
{
    int nx = 3;  // row
    int ny = 2;  // col
    int ns = nx * ny;
    hipfftComplex *d_complex, *complex;

    // Allocate device memory
    complex = static_cast<hipfftComplex*>(malloc(sizeof(hipfftComplex) * ns));
    CHECK_CUDA(hipMalloc(&d_complex, sizeof(hipfftComplex) * ns));

    // Input Generation
    for (int x = 0; x < nx; ++x) {
        for (int y = 0; y < ny; ++y) {
            hipfftComplex c;
            c.x = x + y;
            c.y = 0;
            complex[y + x * ny] = c;
        }
    }
    std::cout << "=== INPUT ===" << std::endl;
    print2D(complex, true, ny, nx, 0, 0, ny, nx);

    // Setup the cuFFT plan
    hipfftHandle plan = 0;
    CHECK_CUFFT(hipfftPlan2d(&plan, nx, ny, HIPFFT_C2C));

    // Transfer inputs into device memory
    CHECK_CUDA(hipMemcpy(d_complex, complex, sizeof(hipfftComplex) * ns, hipMemcpyHostToDevice));

    // warm up
    common::warmUpGpu();

    // Execute a complex-to-complex 1D FFT
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, nullptr));

    CHECK_CUFFT(hipfftExecC2C(plan, d_complex, d_complex, HIPFFT_FORWARD));

    CHECK_CUDA(hipEventRecord(stop, nullptr));
    CHECK_CUDA(hipEventSynchronize(stop););

    float time_ms;
    CHECK_CUDA(hipEventElapsedTime(&time_ms, start, stop));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    ILOG("fft2 {}x{}: {:.3f}ms", nx, ny, time_ms);

    // Retrieve the results into host memory
    CHECK_CUDA(hipMemcpy(complex, d_complex, sizeof(hipfftComplex) * ns, hipMemcpyDeviceToHost));

    std::cout << "=== OUTPUT ===" << std::endl;
    print2D(d_complex, false, ny, nx, 0, 0, ny, nx);

    free(complex);
    CHECK_CUDA(hipFree(d_complex));
    CHECK_CUFFT(hipfftDestroy(plan));

    return MyErrCode::kOk;
}