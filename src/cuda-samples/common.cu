#include "hip/hip_runtime.h"
#include "./common.cuh"

double seconds()
{
    auto now = std::chrono::system_clock::now();
    auto micro = std::chrono::duration_cast<std::chrono::microseconds>(now.time_since_epoch());
    return micro.count() * 1e-6;
}

__global__ void warm_up_gpu()
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid;
}

void warmUpGpu()
{
    warm_up_gpu<<<10 * 1024 * 1024, 1024>>>();
    CHECK(hipDeviceSynchronize());
}